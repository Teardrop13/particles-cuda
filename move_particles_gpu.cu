#include "hip/hip_runtime.h"
#include <math.h>
#include <stdio.h>

#include <cstdlib>
#include <iostream>

#include "move_particles.hpp"

long maxGridSize;
long maxThreadsPerBlock;

int number_of_particles;

float *d_position_x;
float *d_position_y;
float *d_position_z;
float *d_acceleration_x;
float *d_acceleration_y;
float *d_acceleration_z;
float *d_mass;
float *d_step;  // step nie jest do podnoszony do kwadratu ani dzielony przez 2
float *d_number_of_particles;
float *d_G;

__device__ calculate_acceleration_one_particle(float *d_current_position_x,
                                               float *d_current_position_y,
                                               float *d_current_position_z,
                                               float *d_current_acceleration_x,
                                               float *d_current_acceleration_y,
                                               float *d_current_acceleration_z,
                                               float *d_other_position_x,
                                               float *d_other_position_y,
                                               float *d_other_position_z,
                                               float *d_other_mass,
                                               float *d_step,
                                               float *d_G) {

    float distance = pow((*d_current_position_x) - (*d_other_position_x), 2) + pow((*d_current_position_y) - (*d_other_position_y), 2) + pow((*d_current_position_z) - (*d_other_position_z), 2);
    float f = (*d_G) * (*d_other_mass) / distance;

    *d_current_acceleration_x += ((*d_other_position_x) - (*d_current_position_x)) * f;
    *d_current_acceleration_y += ((*d_other_position_y) - (*d_current_position_y)) * f;
    *d_current_acceleration_z += ((*d_other_position_z) - (*d_current_position_z)) * f;
}

__device__ calculate_move_one_particle(float *d_current_position_x,
                                       float *d_current_position_y,
                                       float *d_current_position_z,
                                       float *d_current_acceleration_x,
                                       float *d_current_acceleration_y,
                                       float *d_current_acceleration_z,
                                       float *d_position_x,
                                       float *d_position_y,
                                       float *d_position_z,
                                       float *d_mass,
                                       float *d_step,
                                       int *d_number_of_particles,
                                       float d_G) {

    int i = threadIdx.x + blockIdx.x * blockDim.x;

    <<<1, 1024>>> calculate_acceleration_one_particle(d_current_position_x,
                                               d_current_position_y,
                                               d_current_position_z,
                                               d_current_acceleration_x,
                                               d_current_acceleration_y,
                                               d_current_acceleration_z,
                                               d_position_x[i],
                                               d_position_y[i],
                                               d_position_z[i],
                                               d_mass,
                                               d_time,
                                               d_number_of_particles,
                                               d_G);

    *d_current_position_x += (*d_current_acceleration_x) * (*d_step);
    *d_current_position_y += (*d_current_acceleration_y) * (*d_step);
    *d_current_position_z += (*d_current_acceleration_z) * (*d_step);
}

__global__ void calculate_move_all_particles(float *d_position_x,
                                             float *d_position_y,
                                             float *d_position_z,
                                             float *d_acceleration_x,
                                             float *d_acceleration_y,
                                             float *d_acceleration_z,
                                             float *d_mass,
                                             float *d_step,
                                             int *d_number_of_particles,
                                             float d_G) {

    int i = threadIdx.x + blockIdx.x * blockDim.x;

    <<< 1, 1024>>> calculate_move_one_particle(d_position_x[i],
                                               d_position_y[i],
                                               d_position_z[i],
                                               d_acceleration_x[i],
                                               d_acceleration_y[i],
                                               d_acceleration_z[i],
                                               d_position_x,
                                               d_position_y,
                                               d_position_z,
                                               d_mass,
                                               d_time,
                                               d_number_of_particles,
                                               d_G);
}

void cuda_initialize(float *position_x,
                     float *position_y,
                     float *position_z,
                     float *acceleration_x,
                     float *acceleration_y,
                     float *acceleration_z,
                     float *mass,
                     float *step,
                     int *number_of_particles,
                     float G) {
    int device = 0;

    hipSetDevice(device);
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, device);

    maxGridSize = deviceProp.maxGridSize[0];
    maxThreadsPerBlock = deviceProp.maxThreadsPerBlock;

    std::cout << "==============================================" << std::endl;
    std::cout << "Max dimension size of a grid size (x): " << maxGridSize << std::endl;
    std::cout << "Maximum number of threads per block: " << maxThreadsPerBlock << std::endl;
    std::cout << std::endl;

    hipMalloc((void **)&d_position_x, sizeof(float) * (*number_of_particles));
    hipMemcpy(d_position_x, &position_x, sizeof(float) * (*number_of_particles), hipMemcpyHostToDevice);

    hipMalloc((void **)&d_position_y, sizeof(float) * (*number_of_particles));
    hipMemcpy(d_position_y, &position_y, sizeof(float) * (*number_of_particles), hipMemcpyHostToDevice);

    hipMalloc((void **)&d_position_z, sizeof(float) * (*number_of_particles));
    hipMemcpy(d_position_z, &position_z, sizeof(float) * (*number_of_particles), hipMemcpyHostToDevice);

    hipMalloc((void **)&d_acceleration_x, sizeof(float) * (*number_of_particles));
    hipMemcpy(d_acceleration_x, &acceleration_x, sizeof(float) * (*number_of_particles), hipMemcpyHostToDevice);

    hipFree(d_mass);
    hipMalloc((void **)&d_acceleration_y, sizeof(float) * (*number_of_particles));
    hipMemcpy(d_acceleration_y, &acceleration_y, sizeof(float) * (*number_of_particles), hipMemcpyHostToDevice);

    hipMalloc((void **)&d_acceleration_z, sizeof(float) * (*number_of_particles));
    hipMemcpy(d_acceleration_z, &acceleration_z, sizeof(float) * (*number_of_particles), hipMemcpyHostToDevice);

    hipMalloc((void **)&d_mass, sizeof(float) * (*number_of_particles));
    hipMemcpy(d_mass, &mass, sizeof(float) * (*number_of_particles), hipMemcpyHostToDevice);

    hipMalloc((void **)&d_step, sizeof(float));
    hipMemcpy(d_step, &step, sizeof(float), hipMemcpyHostToDevice);

    hipMalloc((void **)&d_number_of_particles, sizeof(int));
    hipMemcpy(d_number_of_particles, &number_of_particles, sizeof(int), hipMemcpyHostToDevice);

    hipMalloc((void **)&d_G, sizeof(float));
    hipMemcpy(d_G, &G, sizeof(float), hipMemcpyHostToDevice);
}

void cuda_clean() {
    hipFree(d_position_x);
    hipFree(d_position_y);
    hipFree(d_position_z);
    hipFree(d_acceleration_x);
    hipFree(d_acceleration_y);
    hipFree(d_acceleration_z);
    hipFree(d_mass);
    hipFree(d_step);
    hipFree(d_number_of_particles);
    hipFree(d_G);
}

void move_particles(float *position_x,
                    float *position_y,
                    float *position_z) {
    <<< 1, 1024>>> calculate_move_all_particles(d_position_x,
                                                d_position_y,
                                                d_position_z,
                                                d_acceleration_x,
                                                d_acceleration_y,
                                                d_acceleration_z,
                                                d_mass,
                                                d_time,
                                                d_number_of_particles,
                                                d_G);

    hipMemcpy(position_x, &d_position_x, sizeof(float) * number_of_particles, hipMemcpyDeviceToHost);
    hipMemcpy(position_y, &d_position_y, sizeof(float) * number_of_particles, hipMemcpyDeviceToHost);
    hipMemcpy(position_z, &d_position_z, sizeof(float) * number_of_particles, hipMemcpyDeviceToHost);
}