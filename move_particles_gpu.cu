#include "hip/hip_runtime.h"
#include <math.h>
#include <stdio.h>

#include <cstdlib>
#include <iostream>

#include "move_particles_gpu.hpp"

long maxGridSize;
long maxThreadsPerBlock;

int number_of_particles;

int threads;
int *d_threads;
int blocks;
int *d_blocks;

float *d_position_x;
float *d_position_y;
float *d_position_z;
float *d_acceleration_x;
float *d_acceleration_y;
float *d_acceleration_z;
float *d_mass;
float *d_step;  // step nie jest do podnoszony do kwadratu ani dzielony przez 2
int *d_number_of_particles;
float *d_G;

__global__ void calculate_acceleration_one_particle(float *d_current_position_x,
                                               float *d_current_position_y,
                                               float *d_current_position_z,
                                               float *d_current_acceleration_x,
                                               float *d_current_acceleration_y,
                                               float *d_current_acceleration_z,
                                               float *d_other_position_x,
                                               float *d_other_position_y,
                                               float *d_other_position_z,
                                               float *d_other_mass,
                                               float *d_G) {

    float distance = pow((*d_current_position_x) - (*d_other_position_x), 2) + 
    pow((*d_current_position_y) - (*d_other_position_y), 2) + 
    pow((*d_current_position_z) - (*d_other_position_z), 2);
    if (distance == 0) {
        distance = 0.000001;
    }
    float f = (*d_G) * (*d_other_mass) / distance;

    *d_current_acceleration_x += ((*d_other_position_x) - (*d_current_position_x)) * f;
    *d_current_acceleration_y += ((*d_other_position_y) - (*d_current_position_y)) * f;
    *d_current_acceleration_z += ((*d_other_position_z) - (*d_current_position_z)) * f;
}

__global__ void calculate_move_one_particle(float *d_current_position_x,
                                       float *d_current_position_y,
                                       float *d_current_position_z,
                                       float *d_current_acceleration_x,
                                       float *d_current_acceleration_y,
                                       float *d_current_acceleration_z,
                                       float *d_position_x,
                                       float *d_position_y,
                                       float *d_position_z,
                                       float *d_mass,
                                       float *d_step,
                                       float *d_G,
                                       int  *d_number_of_particles,
                                       int *d_blocks,
                                       int *d_threads) {

    int i = threadIdx.x;

    calculate_acceleration_one_particle<<< *d_blocks, *d_threads>>> (d_current_position_x,
                                               d_current_position_y,
                                               d_current_position_z,
                                               d_current_acceleration_x,
                                               d_current_acceleration_y,
                                               d_current_acceleration_z,
                                               &d_position_x[i],
                                               &d_position_y[i],
                                               &d_position_z[i],
                                               d_mass,
                                               d_G);
    __syncthreads();

    *d_current_position_x += (*d_current_acceleration_x) * (*d_step);
    *d_current_position_y += (*d_current_acceleration_y) * (*d_step);
    *d_current_position_z += (*d_current_acceleration_z) * (*d_step);
}

__global__ void calculate_move_all_particles(float *d_position_x,
                                             float *d_position_y,
                                             float *d_position_z,
                                             float *d_acceleration_x,
                                             float *d_acceleration_y,
                                             float *d_acceleration_z,
                                             float *d_mass,
                                             float *d_step,
                                             float *d_G,
                                             int *d_number_of_particles,
                                             int *d_blocks,
                                             int *d_threads) {

    int i = threadIdx.x;

    calculate_move_one_particle<<< *d_blocks, *d_threads>>> (&d_position_x[i],
                                               &d_position_y[i],
                                               &d_position_z[i],
                                               &d_acceleration_x[i],
                                               &d_acceleration_y[i],
                                               &d_acceleration_z[i],
                                               d_position_x,
                                               d_position_y,
                                               d_position_z,
                                               d_mass,
                                               d_step,
                                               d_G,
                                               d_number_of_particles,
                                               d_blocks,
                                               d_threads);

    // d_position_x[i] = d_position_x[i] + 0.05F;
    __syncthreads();
}

void cuda_initialize(float *position_x,
                     float *position_y,
                     float *position_z,
                     float *acceleration_x,
                     float *acceleration_y,
                     float *acceleration_z,
                     float *mass,
                     float step,
                     int _number_of_particles,
                     float G) {
    int device = 0;
    number_of_particles = _number_of_particles;

    blocks = number_of_particles / 16;
    threads = 16;

    hipSetDevice(device);
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, device);

    maxGridSize = deviceProp.maxGridSize[0];
    maxThreadsPerBlock = deviceProp.maxThreadsPerBlock;

    std::cout << "==============================================" << std::endl;
    std::cout << "Max dimension size of a grid size (x): " << maxGridSize << std::endl;
    std::cout << "Maximum number of threads per block: " << maxThreadsPerBlock << std::endl;
    std::cout << std::endl;

    hipMalloc((void **)&d_position_x, sizeof(float) * number_of_particles);
    hipMemcpy(d_position_x, position_x, sizeof(float) * number_of_particles, hipMemcpyHostToDevice);

    hipMalloc((void **)&d_position_y, sizeof(float) * number_of_particles);
    hipMemcpy(d_position_y, position_y, sizeof(float) * number_of_particles, hipMemcpyHostToDevice);

    hipMalloc((void **)&d_position_z, sizeof(float) * number_of_particles);
    hipMemcpy(d_position_z, position_z, sizeof(float) * number_of_particles, hipMemcpyHostToDevice);

    hipMalloc((void **)&d_acceleration_x, sizeof(float) * number_of_particles);
    hipMemcpy(d_acceleration_x, acceleration_x, sizeof(float) * number_of_particles, hipMemcpyHostToDevice);

    hipMalloc((void **)&d_acceleration_y, sizeof(float) * number_of_particles);
    hipMemcpy(d_acceleration_y, acceleration_y, sizeof(float) * number_of_particles, hipMemcpyHostToDevice);

    hipMalloc((void **)&d_acceleration_z, sizeof(float) * number_of_particles);
    hipMemcpy(d_acceleration_z, acceleration_z, sizeof(float) * number_of_particles, hipMemcpyHostToDevice);

    hipMalloc((void **)&d_mass, sizeof(float) * number_of_particles);
    hipMemcpy(d_mass, mass, sizeof(float) * number_of_particles, hipMemcpyHostToDevice);

    hipMalloc((void **)&d_step, sizeof(float));
    hipMemcpy(d_step, &step, sizeof(float), hipMemcpyHostToDevice);

    hipMalloc((void **)&d_number_of_particles, sizeof(int));
    hipMemcpy(d_number_of_particles, &number_of_particles, sizeof(int), hipMemcpyHostToDevice);

    hipMalloc((void **)&d_G, sizeof(float));
    hipMemcpy(d_G, &G, sizeof(float), hipMemcpyHostToDevice);

    hipMalloc((void **)&d_blocks, sizeof(int));
    hipMemcpy(d_blocks, &blocks, sizeof(int), hipMemcpyHostToDevice);

    hipMalloc((void **)&d_threads, sizeof(int));
    hipMemcpy(d_threads, &threads, sizeof(int), hipMemcpyHostToDevice);
}

void cuda_clean() {
    hipFree(d_position_x);
    hipFree(d_position_y);
    hipFree(d_position_z);
    hipFree(d_acceleration_x);
    hipFree(d_acceleration_y);
    hipFree(d_acceleration_z);
    hipFree(d_mass);
    hipFree(d_step);
    hipFree(d_number_of_particles);
    hipFree(d_G);
}

void move_particles(float *position_x,
                    float *position_y,
                    float *position_z) {

    std::cout << "przed: " << position_x[0] << ", " << position_x[number_of_particles-1] << std::endl;

    calculate_move_all_particles<<< blocks, threads>>> (d_position_x,
                                                d_position_y,
                                                d_position_z,
                                                d_acceleration_x,
                                                d_acceleration_y,
                                                d_acceleration_z,
                                                d_mass,
                                                d_step,
                                                d_G,
                                                d_number_of_particles,
                                                d_blocks,
                                                d_threads);

    hipDeviceSynchronize();

    hipMemcpy(position_x, d_position_x, sizeof(float) * number_of_particles, hipMemcpyDeviceToHost);
    hipMemcpy(position_y, d_position_y, sizeof(float) * number_of_particles, hipMemcpyDeviceToHost);
    hipMemcpy(position_z, d_position_z, sizeof(float) * number_of_particles, hipMemcpyDeviceToHost);
    std::cout << "po skopiowaniu: " << position_x[0] << ", " << position_x[number_of_particles-1] << std::endl;
}