#include "hip/hip_runtime.h"
#include <math.h>
#include <stdio.h>

#include <cstdlib>
#include <iostream>

#include "move_particles.hpp"


long maxGridSize;
long maxThreadsPerBlock;

int length;

float *d_position_x;
float *d_position_y;
float *d_position_z;
float *d_acceleration_x;
float *d_acceleration_y;
float *d_acceleration_z;
float *d_mass;
float *d_step;
float *d_length;
float *d_G;

__global__ void calculate_move(float *d_position_x,
                               float *d_position_y,
                               float *d_position_z,
                               float *d_acceleration_x,
                               float *d_acceleration_y,
                               float *d_acceleration_z,
                               float *d_mass,
                               float *d_step,
                               int *d_length,
                               float d_G) {
    for (int i = 0; i < *d_length; i++) {
        for (int k = 0; k < *d_length; k++) {
            float distance = pow(d_position_x[i] - d_position_x[k], 2) + pow(d_position_y[i] - position_y[k], 2) + pow(position_z[i] - position_z[k], 2);
            if (distance == 0) {
                distance = 0.000000001;
            }

            float f = G * mass[k] / distance;
            acceleration_x[i] += (position_x[k] - position_x[i]) * f;
            acceleration_y[i] += (position_y[k] - position_y[i]) * f;
            acceleration_z[i] += (position_z[k] - position_z[i]) * f;
        }
    }

    for (int i = 0; i < length; i++) {
        position_x[i] += acceleration_x[i] * time / 2;
        position_y[i] += acceleration_y[i] * time / 2;
        position_z[i] += acceleration_z[i] * time / 2;
    }
}

void cuda_initialize(float *position_x,
                     float *position_y,
                     float *position_z,
                     float *acceleration_x,
                     float *acceleration_y,
                     float *acceleration_z,
                     float *mass,
                     float *step,
                     int *length,
                     float G) {
    int device = 0;

    hipSetDevice(device);
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, device);

    maxGridSize = deviceProp.maxGridSize[0];
    maxThreadsPerBlock = deviceProp.maxThreadsPerBlock;

    std::cout << "==============================================" << std::endl;
    std::cout << "Max dimension size of a grid size (x): " << maxGridSize << std::endl;
    std::cout << "Maximum number of threads per block: " << maxThreadsPerBlock << std::endl;
    std::cout << std::endl;

    hipMalloc((void **)&d_position_x, sizeof(float) * (*length));
    hipMemcpy(d_position_x, &position_x, sizeof(float) * (*length), hipMemcpyHostToDevice);

    hipMalloc((void **)&d_position_y, sizeof(float) * (*length));
    hipMemcpy(d_position_y, &position_y, sizeof(float) * (*length), hipMemcpyHostToDevice);

    hipMalloc((void **)&d_position_z, sizeof(float) * (*length));
    hipMemcpy(d_position_z, &position_z, sizeof(float) * (*length), hipMemcpyHostToDevice);

    hipMalloc((void **)&d_acceleration_x, sizeof(float) * (*length));
    hipMemcpy(d_acceleration_x, &acceleration_x, sizeof(float) * (*length), hipMemcpyHostToDevice);

    hipFree(d_mass);
    hipMalloc((void **)&d_acceleration_y, sizeof(float) * (*length));
    hipMemcpy(d_acceleration_y, &acceleration_y, sizeof(float) * (*length), hipMemcpyHostToDevice);

    hipMalloc((void **)&d_acceleration_z, sizeof(float) * (*length));
    hipMemcpy(d_acceleration_z, &acceleration_z, sizeof(float) * (*length), hipMemcpyHostToDevice);

    hipMalloc((void **)&d_mass, sizeof(float) * (*length));
    hipMemcpy(d_mass, &mass, sizeof(float) * (*length), hipMemcpyHostToDevice);

    hipMalloc((void **)&d_step, sizeof(float));
    hipMemcpy(d_step, &step, sizeof(float), hipMemcpyHostToDevice);

    hipMalloc((void **)&d_length, sizeof(int));
    hipMemcpy(d_length, &length, sizeof(int), hipMemcpyHostToDevice);

    hipMalloc((void **)&d_G, sizeof(float));
    hipMemcpy(d_G, &G, sizeof(float), hipMemcpyHostToDevice);
}

void cuda_clean() {
    hipFree(d_position_x);
    hipFree(d_position_y);
    hipFree(d_position_z);
    hipFree(d_acceleration_x);
    hipFree(d_acceleration_y);
    hipFree(d_acceleration_z);
    hipFree(d_mass);
    hipFree(d_step);
    hipFree(d_length);
    hipFree(d_G);
}

void move_particles(float *position_x,
                    float *position_y,
                    float *position_z) {

    <<<>>> calculate_move(d_position_x,
                          d_position_y,
                          d_position_z,
                          d_acceleration_x,
                          d_acceleration_y,
                          d_acceleration_z,
                          d_mass,
                          d_time,
                          d_length,
                          d_G);

    hipMemcpy(position_x, &d_position_x, sizeof(float) * length, hipMemcpyDeviceToHost);
    hipMemcpy(position_y, &d_position_y, sizeof(float) * length, hipMemcpyDeviceToHost);
    hipMemcpy(position_z, &d_position_z, sizeof(float) * length, hipMemcpyDeviceToHost);
}