#include "hip/hip_runtime.h"
#include <math.h>
#include <stdio.h>

#include <cstdlib>
#include <iostream>

#include "move_particles_gpu.hpp"

long maxGridSize;
long maxThreadsPerBlock;

int number_of_particles;

int threads;
int *d_threads;
int blocks;
int *d_blocks;

Particle *d_particles;
float *d_dt;
float *d_G;

#define cuda_check(ans) \
    { _check((ans), __LINE__); }
inline void _check(hipError_t code, int line) {
    if (code != hipSuccess) {
        fprintf(stderr, "CUDA Error:\n%s\n%d\n", hipGetErrorString(code), line);
        exit(code);
    }
}

__device__ float get_distance(Vector a, Vector b) {
    return sqrt(pow(a.x - b.x, 2) + pow(a.y - b.y, 2) + pow(a.z - b.z, 2));
}



__global__ void calculate_speed_all_to_one_particle(Particle *current_particle,
                                                    Particle *d_particles,
                                                    float *d_dt,
                                                    float *d_G) {

    int i = threadIdx.x + blockIdx.x * blockDim.x / 3;
    int coordinate = (threadIdx.x + blockIdx.x * blockDim.x) % 3;

    float distance = get_distance(d_particles[i].position, d_particles[i].position);

    if (distance < 0.001) {
        return;
    }


    float a = (*d_G) * d_particles[i].mass / pow(distance, 3);

    if (coordinate == 0) {
        atomicAdd(&(*current_particle).speed.x, (d_particles[i].position.x - d_particles[i].position.x) * a * (*d_dt));
    } if (coordinate == 1) {
        atomicAdd(&(*current_particle).speed.y, (d_particles[i].position.y - d_particles[i].position.y) * a * (*d_dt));
    } else {
        atomicAdd(&(*current_particle).speed.z, (d_particles[i].position.z - d_particles[i].position.z) * a * (*d_dt));
    }
}

__global__ void calculate_speed_all_to_all_particles(Particle *d_particles,
                                                     float *d_dt,
                                                     float *d_G,
                                                     int *d_blocks,
                                                     int *d_threads) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;

    // resetowanie prędkości
    // d_particles[i].speed = Vector(0,0,0);
    calculate_speed_all_to_one_particle<<<*d_blocks * 3, *d_threads>>>(&d_particles[i],
                                                                   d_particles,
                                                                   d_dt,
                                                                   d_G);
}

__global__ void calculate_position_all_particles(Particle *d_particles, float *d_dt) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;

    d_particles[i].position += (d_particles[i].speed * Vector(*d_dt,*d_dt,*d_dt));
    // d_particles[i].position += d_particles[i].speed;
    // d_particles[i].position += (d_particles[i].speed * (*d_dt));
    // d_particles[i].position.x += 2.;
}

void cuda_initialize(Particle *particles,
                     int _number_of_particles,
                     float dt,
                     float G) {
    int device = 0;
    number_of_particles = _number_of_particles;

    threads = 64;
    blocks = number_of_particles / threads;

    // threads = 32;
    // blocks = 32;

    hipSetDevice(device);
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, device);

    maxGridSize = deviceProp.maxGridSize[0];
    maxThreadsPerBlock = deviceProp.maxThreadsPerBlock;

    std::cout << "==============================================" << std::endl;
    std::cout << "Max dimension size of a grid size (x): " << maxGridSize << std::endl;
    std::cout << "Maximum number of threads per block: " << maxThreadsPerBlock << std::endl;
    std::cout << "==============================================" << std::endl;
    std::cout << std::endl;

    hipMalloc((void **)&d_particles, sizeof(Particle) * number_of_particles);
    hipMemcpy(d_particles, particles, sizeof(Particle) * number_of_particles, hipMemcpyHostToDevice);

    hipMalloc((void **)&d_dt, sizeof(float));
    hipMemcpy(d_dt, &dt, sizeof(float), hipMemcpyHostToDevice);

    hipMalloc((void **)&d_G, sizeof(float));
    hipMemcpy(d_G, &G, sizeof(float), hipMemcpyHostToDevice);

    hipMalloc((void **)&d_blocks, sizeof(int));
    hipMemcpy(d_blocks, &blocks, sizeof(int), hipMemcpyHostToDevice);

    hipMalloc((void **)&d_threads, sizeof(int));
    hipMemcpy(d_threads, &threads, sizeof(int), hipMemcpyHostToDevice);
}

void cuda_clean() {
    hipFree(d_particles);
    hipFree(d_dt);
    hipFree(d_G);
    hipFree(d_blocks);
    hipFree(d_threads);
}

void move_particles(Particle *particles) {
    // std::cout << "gpu przed: " << particles[0].speed.x << std::endl;

    float time;
    hipEvent_t start, stop;

    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);

    calculate_speed_all_to_all_particles<<<blocks, threads>>>(d_particles,
                                                       d_dt,
                                                       d_G,
                                                       d_blocks,
                                                       d_threads);

    hipDeviceSynchronize();

    calculate_position_all_particles<<<blocks, threads>>>(d_particles, d_dt);

    hipDeviceSynchronize();

    cuda_check(hipMemcpy(particles, d_particles, sizeof(Particle) * number_of_particles, hipMemcpyDeviceToHost));

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time, start, stop);

    std::cout << time << std::endl;

    // std::cout << "gpu po: " << particles[0].speed.x << std::endl;
}