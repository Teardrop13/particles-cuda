#include "hip/hip_runtime.h"
#include <math.h>
#include <stdio.h>

#include <cstdlib>
#include <iostream>

#include "move_particles.hpp"

const float G = 2;  // nie prawdziwe G

long maxGridSize;
long maxThreadsPerBlock;

float *position_x;
float *position_y;
float *position_z;
float *acceleration_x;
float *acceleration_y;
float *acceleration_z;
int length;
float *mass;

float *d_position_x;
float *d_position_y;
float *d_position_z;
float *d_acceleration_x;
float *d_acceleration_y;
float *d_acceleration_z;
int *d_length;
float *d_mass;

__global__ void calculate_move(float *position_x,
                               float *position_y,
                               float *position_z,
                               float *acceleration_x,
                               float *acceleration_y,
                               float *acceleration_z,
                               float *mass,
                               int length) {
    for (int i = 0; i < length; i++) {
        for (int k = 0; k < length; k++) {
            float distance = pow(position_x[i] - position_x[k], 2) + pow(position_y[i] - position_y[k], 2) + pow(position_z[i] - position_z[k], 2);
            if (distance == 0) {
                distance = 0.000000001;
            }

            float f = G * mass / distance;
            acceleration_x[i] += (position_x[k] - position_x[i]) * f;
            acceleration_y[i] += (position_y[k] - position_y[i]) * f;
            acceleration_z[i] += (position_z[k] - position_z[i]) * f;
        }
    }

    for (int i = 0; i < length; i++) {
        position_x[i] += acceleration_x[i] * time / 2;
        position_y[i] += acceleration_y[i] * time / 2;
        position_z[i] += acceleration_z[i] * time / 2;
    }
}

void cuda_initialize(float *position_x,
                     float *position_y,
                     float *position_z,
                     float *acceleration_x,
                     float *acceleration_y,
                     float *acceleration_z,
                     float *mass,
                     int length) {
    int device = 0;

    hipSetDevice(device);
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, device);

    maxGridSize = deviceProp.maxGridSize[0];
    maxThreadsPerBlock = deviceProp.maxThreadsPerBlock;

    std::cout << "==============================================" << std::endl;
    std::cout << "Max dimension size of a grid size (x): " << maxGridSize << std::endl;
    std::cout << "Maximum number of threads per block: " << maxThreadsPerBlock << std::endl;
    std::cout << std::endl;

    hipMalloc((void **)&d_position_x, sizeof(float) * length);
    hipMemcpy(d_position_x, &position_x, sizeof(float) * length, hipMemcpyHostToDevice);

    hipMalloc((void **)&d_position_y, sizeof(float) * length);
    hipMemcpy(d_position_y, &position_y, sizeof(float) * length, hipMemcpyHostToDevice);

    hipMalloc((void **)&d_position_z, sizeof(float) * length);
    hipMemcpy(d_position_z, &position_z, sizeof(float) * length, hipMemcpyHostToDevice);

    hipMalloc((void **)&d_acceleration_x, sizeof(float) * length);
    hipMemcpy(d_acceleration_x, &acceleration_x, sizeof(float) * length, hipMemcpyHostToDevice);

    hipMalloc((void **)&d_acceleration_y, sizeof(float) * length);
    hipMemcpy(d_acceleration_y, &acceleration_y, sizeof(float) * length, hipMemcpyHostToDevice);

    hipMalloc((void **)&d_acceleration_z, sizeof(float) * length);
    hipMemcpy(d_acceleration_z, &acceleration_z, sizeof(float) * length, hipMemcpyHostToDevice);

    hipMalloc((void **)&d_mass, sizeof(float) * length);
    hipMemcpy(d_mass, &mass, sizeof(float) * length, hipMemcpyHostToDevice);

    hipMalloc((void **)&d_length, sizeof(int));
    hipMemcpy(d_length, &length, sizeof(int), hipMemcpyHostToDevice);
}

void cuda_clean() {
    hipFree(d_position_x);
    hipFree(d_position_y);
    hipFree(d_position_z);
    hipFree(d_acceleration_x);
    hipFree(d_acceleration_y);
    hipFree(d_acceleration_z);
    hipFree(d_mass);
    hipFree(d_length);
}

void move_particles(float *position_x,
                    float *position_y,
                    float *position_z) {
    float time = 0.1;
    time = pow(time, 2);

    float mass = 2;

    <<<>>> calculate_move(d_position_x,
                          d_position_y,
                          d_position_z,
                          d_acceleration_x,
                          d_acceleration_y,
                          d_acceleration_z,
                          d_mass,
                          d_length);

    hipMemcpy(position_x, &d_position_x, sizeof(float) * length, hipMemcpyDeviceToHost);
    hipMemcpy(position_y, &d_position_y, sizeof(float) * length, hipMemcpyDeviceToHost);
    hipMemcpy(position_z, &d_position_z, sizeof(float) * length, hipMemcpyDeviceToHost);
}