#include "hip/hip_runtime.h"
#include <math.h>
#include <stdio.h>

#include <cstdlib>
#include <iostream>

#include "move_particles_gpu.hpp"

long maxGridSize;
long maxThreadsPerBlock;

int number_of_particles;

float *d_position_x;
float *d_position_y;
float *d_position_z;
float *d_acceleration_x;
float *d_acceleration_y;
float *d_acceleration_z;
float *d_mass;
float *d_step;  // step nie jest do podnoszony do kwadratu ani dzielony przez 2
float *d_number_of_particles;
float *d_G;

__global__ void calculate_acceleration_one_particle(float *d_current_position_x,
                                               float *d_current_position_y,
                                               float *d_current_position_z,
                                               float *d_current_acceleration_x,
                                               float *d_current_acceleration_y,
                                               float *d_current_acceleration_z,
                                               float *d_other_position_x,
                                               float *d_other_position_y,
                                               float *d_other_position_z,
                                               float *d_other_mass,
                                               float *d_G) {

    float distance = pow((*d_current_position_x) - (*d_other_position_x), 2) + 
    pow((*d_current_position_y) - (*d_other_position_y), 2) + 
    pow((*d_current_position_z) - (*d_other_position_z), 2);
    if (distance == 0) {
        distance = 0.000001;
    }
    float f = (*d_G) * (*d_other_mass) / distance;

    *d_current_acceleration_x += ((*d_other_position_x) - (*d_current_position_x)) * f;
    *d_current_acceleration_y += ((*d_other_position_y) - (*d_current_position_y)) * f;
    *d_current_acceleration_z += ((*d_other_position_z) - (*d_current_position_z)) * f;
}

__global__ void calculate_move_one_particle(float *d_current_position_x,
                                       float *d_current_position_y,
                                       float *d_current_position_z,
                                       float *d_current_acceleration_x,
                                       float *d_current_acceleration_y,
                                       float *d_current_acceleration_z,
                                       float *d_position_x,
                                       float *d_position_y,
                                       float *d_position_z,
                                       float *d_mass,
                                       float *d_step,
                                       float *d_G) {

    int i = threadIdx.x + blockIdx.x * blockDim.x;

    calculate_acceleration_one_particle<<< 1, 1024>>> (d_current_position_x,
                                               d_current_position_y,
                                               d_current_position_z,
                                               d_current_acceleration_x,
                                               d_current_acceleration_y,
                                               d_current_acceleration_z,
                                               &d_position_x[i],
                                               &d_position_y[i],
                                               &d_position_z[i],
                                               d_mass,
                                               d_G);

    *d_current_position_x += (*d_current_acceleration_x) * (*d_step);
    *d_current_position_y += (*d_current_acceleration_y) * (*d_step);
    *d_current_position_z += (*d_current_acceleration_z) * (*d_step);
}

__global__ void calculate_move_all_particles(float *d_position_x,
                                             float *d_position_y,
                                             float *d_position_z,
                                             float *d_acceleration_x,
                                             float *d_acceleration_y,
                                             float *d_acceleration_z,
                                             float *d_mass,
                                             float *d_step,
                                             float *d_G) {

    int i = threadIdx.x + blockIdx.x * blockDim.x;

    calculate_move_one_particle<<< 1, 1024>>> (&d_position_x[i],
                                               &d_position_y[i],
                                               &d_position_z[i],
                                               &d_acceleration_x[i],
                                               &d_acceleration_y[i],
                                               &d_acceleration_z[i],
                                               d_position_x,
                                               d_position_y,
                                               d_position_z,
                                               d_mass,
                                               d_step,
                                               d_G);
}

void cuda_initialize(float *position_x,
                     float *position_y,
                     float *position_z,
                     float *acceleration_x,
                     float *acceleration_y,
                     float *acceleration_z,
                     float *mass,
                     float step,
                     int _number_of_particles,
                     float G) {
    int device = 0;
    number_of_particles = _number_of_particles;

    hipSetDevice(device);
    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, device);

    maxGridSize = deviceProp.maxGridSize[0];
    maxThreadsPerBlock = deviceProp.maxThreadsPerBlock;

    std::cout << "==============================================" << std::endl;
    std::cout << "Max dimension size of a grid size (x): " << maxGridSize << std::endl;
    std::cout << "Maximum number of threads per block: " << maxThreadsPerBlock << std::endl;
    std::cout << std::endl;

    hipMalloc((void **)&d_position_x, sizeof(float) * number_of_particles);
    hipMemcpy(d_position_x, &position_x, sizeof(float) * number_of_particles, hipMemcpyHostToDevice);

    hipMalloc((void **)&d_position_y, sizeof(float) * number_of_particles);
    hipMemcpy(d_position_y, &position_y, sizeof(float) * number_of_particles, hipMemcpyHostToDevice);

    hipMalloc((void **)&d_position_z, sizeof(float) * number_of_particles);
    hipMemcpy(d_position_z, &position_z, sizeof(float) * number_of_particles, hipMemcpyHostToDevice);

    hipMalloc((void **)&d_acceleration_x, sizeof(float) * number_of_particles);
    hipMemcpy(d_acceleration_x, &acceleration_x, sizeof(float) * number_of_particles, hipMemcpyHostToDevice);

    hipMalloc((void **)&d_acceleration_y, sizeof(float) * number_of_particles);
    hipMemcpy(d_acceleration_y, &acceleration_y, sizeof(float) * number_of_particles, hipMemcpyHostToDevice);

    hipMalloc((void **)&d_acceleration_z, sizeof(float) * number_of_particles);
    hipMemcpy(d_acceleration_z, &acceleration_z, sizeof(float) * number_of_particles, hipMemcpyHostToDevice);

    hipMalloc((void **)&d_mass, sizeof(float) * number_of_particles);
    hipMemcpy(d_mass, &mass, sizeof(float) * number_of_particles, hipMemcpyHostToDevice);

    hipMalloc((void **)&d_step, sizeof(float));
    hipMemcpy(d_step, &step, sizeof(float), hipMemcpyHostToDevice);

    hipMalloc((void **)&d_number_of_particles, sizeof(int));
    hipMemcpy(d_number_of_particles, &number_of_particles, sizeof(int), hipMemcpyHostToDevice);

    hipMalloc((void **)&d_G, sizeof(float));
    hipMemcpy(d_G, &G, sizeof(float), hipMemcpyHostToDevice);
}

void cuda_clean() {
    hipFree(d_position_x);
    hipFree(d_position_y);
    hipFree(d_position_z);
    hipFree(d_acceleration_x);
    hipFree(d_acceleration_y);
    hipFree(d_acceleration_z);
    hipFree(d_mass);
    hipFree(d_step);
    hipFree(d_number_of_particles);
    hipFree(d_G);
}

void move_particles(float *position_x,
                    float *position_y,
                    float *position_z) {
    calculate_move_all_particles<<< 1, 1024>>> (d_position_x,
                                                d_position_y,
                                                d_position_z,
                                                d_acceleration_x,
                                                d_acceleration_y,
                                                d_acceleration_z,
                                                d_mass,
                                                d_step,
                                                d_G);

    hipMemcpy(position_x, &d_position_x, sizeof(float) * number_of_particles, hipMemcpyDeviceToHost);
    hipMemcpy(position_y, &d_position_y, sizeof(float) * number_of_particles, hipMemcpyDeviceToHost);
    hipMemcpy(position_z, &d_position_z, sizeof(float) * number_of_particles, hipMemcpyDeviceToHost);
}